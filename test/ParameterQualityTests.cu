#include "hip/hip_runtime.h"
#include "PrefixTree.h"
#include "RandomnessTest.h"
#include "shuffle/FeistelBijectiveShuffle.h"
#include <condition_variable>
#include <gtest/gtest.h>
#include <thread>

template <class ShuffleType>
class ParameterQualityTests : public RandomnessTests<ShuffleType>
{
};

template <uint64_t NumRounds>
using ParamFeistelBijectiveScanShuffle =
    BijectiveFunctionScanShuffle<FeistelBijectiveFunction<NumRounds>, thrust::host_vector<uint64_t>, DefaultRandomGenerator>;

template <uint64_t NumRounds, class RoundFunction>
using ParamRoundFeistelBijectiveScanShuffle =
    BijectiveFunctionScanShuffle<FeistelBijectiveFunction<NumRounds, RoundFunction>, thrust::host_vector<uint64_t>, DefaultRandomGenerator>;

//using ParameterQualityShuffleTypes = ::testing::Types<StdShuffle<thrust::host_vector<uint64_t>>,
//                                                      ParamFeistelBijectiveScanShuffle<1>,
//                                                      ParamFeistelBijectiveScanShuffle<2>,
//                                                      ParamFeistelBijectiveScanShuffle<3>,
//                                                      ParamFeistelBijectiveScanShuffle<4>,
//                                                      ParamFeistelBijectiveScanShuffle<5>,
//                                                      ParamFeistelBijectiveScanShuffle<6>,
//                                                      ParamFeistelBijectiveScanShuffle<7>,
//                                                      ParamFeistelBijectiveScanShuffle<8>,
//                                                      ParamFeistelBijectiveScanShuffle<9>,
//                                                      ParamFeistelBijectiveScanShuffle<10>,
//                                                      ParamFeistelBijectiveScanShuffle<11>,
//                                                      ParamFeistelBijectiveScanShuffle<12>,
//                                                      ParamFeistelBijectiveScanShuffle<13>,
//                                                      ParamFeistelBijectiveScanShuffle<14>,
//                                                      ParamFeistelBijectiveScanShuffle<15>,
//                                                      ParamFeistelBijectiveScanShuffle<16>,
//                                                      ParamFeistelBijectiveScanShuffle<17>,
//                                                      ParamFeistelBijectiveScanShuffle<18>,
//                                                      ParamFeistelBijectiveScanShuffle<19>,
//                                                      ParamFeistelBijectiveScanShuffle<20>,
//                                                      ParamFeistelBijectiveScanShuffle<21>,
//                                                      ParamFeistelBijectiveScanShuffle<22>,
//                                                      ParamFeistelBijectiveScanShuffle<23>,
//                                                      ParamFeistelBijectiveScanShuffle<24>,
//                                                      ParamFeistelBijectiveScanShuffle<25>,
//                                                      ParamFeistelBijectiveScanShuffle<26>,
//                                                      ParamFeistelBijectiveScanShuffle<27>,
//                                                      ParamFeistelBijectiveScanShuffle<28>,
//                                                      ParamFeistelBijectiveScanShuffle<29>,
//                                                      ParamFeistelBijectiveScanShuffle<30>,
//                                                      ParamFeistelBijectiveScanShuffle<31>,
//                                                      ParamFeistelBijectiveScanShuffle<32>>;

constexpr uint64_t target_num_rounds = 16;
using ParameterQualityShuffleTypes = ::testing::Types<
    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Taus88RanluxRoundFunction<target_num_rounds>>,
    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Taus88LCGRoundFunction<target_num_rounds>>,
    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, RanluxLCGRoundFunction<target_num_rounds>>,
//    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Taus88RoundFunction<target_num_rounds>>,
//    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, LCGRoundFunction<target_num_rounds>>,
//    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Ranlux24RoundFunction<target_num_rounds>>,
//    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Ranlux48RoundFunction<target_num_rounds>>,
    ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, WyHashRoundFunction<target_num_rounds>>,
    StdShuffle<thrust::host_vector<uint64_t>>>;

TYPED_TEST_SUITE( ParameterQualityTests, ParameterQualityShuffleTypes );

uint64_t factorial( uint64_t num )
{
    uint64_t res = 1;
    for( uint64_t i = 1; i <= num; i++ )
        res *= i;
    return res;
}

template <class Vector>
uint64_t permutationToIndex( const Vector& permutation, uint64_t size )
{
    uint64_t res = 0;
    uint64_t base = 1;
    // Interpret the permutation as a number with base shuffle_size
    for( uint64_t i = 0; i < size; i++ )
    {
        res += permutation[i] * base;
        base *= size;
    }
    return res;
}

const std::vector<uint64_t>& allPermutations( uint64_t size )
{
    static std::vector<uint64_t> all;
    if( all.size() != size )
    {
        const uint64_t size_fact = factorial( size );
        all.resize( size_fact );
        all.shrink_to_fit();
        std::vector<uint64_t> items( size );
        std::iota( items.begin(), items.end(), 0 );
        for( uint64_t i = 0; i < size_fact; i++ )
        {
            all[i] = permutationToIndex( items, size );
            std::next_permutation( items.begin(), items.end() );
        }
    }
    return all;
}

void reportStats( std::vector<double>& scores )
{
    std::sort( scores.begin(), scores.end() );
    double sum = std::accumulate( scores.begin(), scores.end(), 0.0 );
    double min = scores.front();
    double max = scores.back();
    double median = ( scores[scores.size() / 2] + scores[( scores.size() + 1 ) / 2] ) / 2;
    double lquart = ( scores[( scores.size() ) / 4] + scores[( scores.size() + 3 ) / 4] ) / 2;
    double uquart = ( scores[( scores.size() * 3 ) / 4] + scores[( scores.size() * 3 + 3 ) / 4] ) / 2;
    std::cout << "Min: " << min << ", LQ: " << lquart << ", Median: " << median
              << ", UQ: " << uquart << ", Max: " << max;
    std::cout << ", Mean: " << sum / (double)scores.size() << std::endl;
}

TYPED_TEST( ParameterQualityTests, FullPermutation )
{
    const uint64_t num_loops = 500;
    const uint64_t seed_start = 0xdeadbeef;
    std::vector<double> p_scores;
    for( uint64_t loop = 0; loop < num_loops; loop++ )
    {
        const uint64_t shuffle_size = 6;
        const uint64_t num_samples = 1e6;

        const uint64_t num_threads = 6;
        const uint64_t samples_per_thread = ( num_samples + ( num_threads - 1 ) ) / num_threads;

        std::vector<std::unordered_map<uint64_t, uint64_t>> results_map( num_threads );
        std::vector<std::thread> threads;
        for( uint64_t tid = 0; tid < num_threads; tid++ )
        {
            threads.emplace_back( [&, tid]() {
                auto local_shuffle = this->shuffle;
                thrust::host_vector<uint64_t> input( shuffle_size );
                thrust::host_vector<uint64_t> output( shuffle_size );

                for( uint64_t i = tid * samples_per_thread;
                     i < std::min( num_samples, samples_per_thread * ( tid + 1 ) ); i++ )
                {
                    thrust::sequence( input.begin(), input.end(), 0 );
                    local_shuffle( input, output, seed_start + loop * num_samples + i, shuffle_size );
                    const uint64_t index = permutationToIndex( output, shuffle_size );
                    results_map[tid][index]++;
                }
            } );
        }

        std::unordered_map<uint64_t, uint64_t> results;
        for( auto& thread : threads )
            thread.join();

        for( auto& res : results_map )
            for( auto& pair : res )
                results[pair.first] += pair.second;

        const uint64_t size_fact = factorial( shuffle_size );
        const double expected_occurances = num_samples / (double)size_fact;

        auto& permutations = allPermutations( shuffle_size );
        double chi_squared = 0.0;
        for( uint64_t i = 0; i < size_fact; i++ )
        {
            chi_squared += pow( results[permutations[i]] - expected_occurances, 2 ) / expected_occurances;
        }

        double p_score = cephes_igamc( (double)( size_fact - 1 ) / 2.0, chi_squared / 2.0 );
        std::cout << p_score << ',' << std::flush;

        p_scores.emplace_back( p_score );
    }

    std::cout << std::endl;
    reportStats( p_scores );
}