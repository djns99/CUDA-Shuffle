#include "hip/hip_runtime.h"
#include "PrefixTree.h"
#include "RandomnessTest.h"
#include "shuffle/FeistelBijectiveShuffle.h"
#include <gtest/gtest.h>
#include <thread>
#include <unordered_map>

template <class ShuffleType>
class ParameterQualityTests : public RandomnessTests<ShuffleType>
{
};

constexpr bool USE_PVALUE = true;
constexpr uint64_t NUM_SAMPLES = 500;
constexpr uint64_t SAMPLE_SIZE = 1e6;

template <uint64_t NumRounds>
using ParamFeistelBijectiveScanShuffle =
    BijectiveFunctionScanShuffle<FeistelBijectiveFunction<NumRounds, WyHashRoundFunction<NumRounds>>, thrust::host_vector<uint64_t>, DefaultRandomGenerator>;

template <uint64_t NumRounds, class RoundFunction>
using ParamRoundFeistelBijectiveScanShuffle =
    BijectiveFunctionScanShuffle<FeistelBijectiveFunction<NumRounds, RoundFunction>, thrust::host_vector<uint64_t>, DefaultRandomGenerator>;

/*
 using ParameterQualityShuffleTypes = ::testing::Types<StdShuffle<thrust::host_vector<uint64_t>>,
                                                      ParamFeistelBijectiveScanShuffle<1>,
                                                      ParamFeistelBijectiveScanShuffle<2>,
                                                      ParamFeistelBijectiveScanShuffle<3>,
                                                      ParamFeistelBijectiveScanShuffle<4>,
                                                      ParamFeistelBijectiveScanShuffle<5>,
                                                      ParamFeistelBijectiveScanShuffle<6>,
                                                      ParamFeistelBijectiveScanShuffle<7>,
                                                      ParamFeistelBijectiveScanShuffle<8>,
                                                      ParamFeistelBijectiveScanShuffle<9>,
                                                      ParamFeistelBijectiveScanShuffle<10>,
                                                      ParamFeistelBijectiveScanShuffle<11>,
                                                      ParamFeistelBijectiveScanShuffle<12>,
                                                      ParamFeistelBijectiveScanShuffle<13>,
                                                      ParamFeistelBijectiveScanShuffle<14>,
                                                      ParamFeistelBijectiveScanShuffle<15>,
                                                      ParamFeistelBijectiveScanShuffle<16>,
                                                      ParamFeistelBijectiveScanShuffle<17>,
                                                      ParamFeistelBijectiveScanShuffle<18>,
                                                      ParamFeistelBijectiveScanShuffle<19>,
                                                      ParamFeistelBijectiveScanShuffle<20>,
                                                      ParamFeistelBijectiveScanShuffle<21>,
                                                      ParamFeistelBijectiveScanShuffle<22>,
                                                      ParamFeistelBijectiveScanShuffle<23>,
                                                      ParamFeistelBijectiveScanShuffle<24>,
                                                      ParamFeistelBijectiveScanShuffle<25>,
                                                      ParamFeistelBijectiveScanShuffle<26>,
                                                      ParamFeistelBijectiveScanShuffle<27>,
                                                      ParamFeistelBijectiveScanShuffle<28>,
                                                      ParamFeistelBijectiveScanShuffle<29>,
                                                      ParamFeistelBijectiveScanShuffle<30>,
                                                      ParamFeistelBijectiveScanShuffle<31>,
                                                      ParamFeistelBijectiveScanShuffle<32>

                                                      >;
*/

constexpr uint64_t target_num_rounds = 16;
using ParameterQualityShuffleTypes =
    ::testing::Types<ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Taus88RanluxRoundFunction<target_num_rounds>>,
                     ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, Taus88LCGRoundFunction<target_num_rounds>>,
                     ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, RanluxLCGRoundFunction<target_num_rounds>>,
                     ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, WyHashRoundFunction<target_num_rounds>>,
                     ParamRoundFeistelBijectiveScanShuffle<target_num_rounds, RC5RoundFunction<target_num_rounds>>,
                     StdShuffle<thrust::host_vector<uint64_t>>>;

TYPED_TEST_SUITE( ParameterQualityTests, ParameterQualityShuffleTypes );

uint64_t factorial( uint64_t num )
{
    uint64_t res = 1;
    for( uint64_t i = 1; i <= num; i++ )
        res *= i;
    return res;
}

template <class Vector>
uint64_t permutationToIndex( const Vector& permutation, uint64_t size )
{
    uint64_t res = 0;
    uint64_t base = 1;
    // Interpret the permutation as a number with base shuffle_size
    for( uint64_t i = 0; i < size; i++ )
    {
        res += permutation[i] * base;
        base *= size;
    }
    return res;
}

const std::vector<uint64_t>& allPermutations( uint64_t size )
{
    static std::vector<uint64_t> all;
    if( all.size() != size )
    {
        const uint64_t size_fact = factorial( size );
        all.resize( size_fact );
        all.shrink_to_fit();
        std::vector<uint64_t> items( size );
        std::iota( items.begin(), items.end(), 0 );
        for( uint64_t i = 0; i < size_fact; i++ )
        {
            all[i] = permutationToIndex( items, size );
            std::next_permutation( items.begin(), items.end() );
        }
    }
    return all;
}

void reportStats( std::vector<double>& scores )
{
    std::sort( scores.begin(), scores.end() );
    double sum = std::accumulate( scores.begin(), scores.end(), 0.0 );
    double min = scores.front();
    double max = scores.back();
    double median = ( scores[scores.size() / 2] + scores[( scores.size() + 1 ) / 2] ) / 2;
    double lquart = ( scores[( scores.size() ) / 4] + scores[( scores.size() + 3 ) / 4] ) / 2;
    double uquart = ( scores[( scores.size() * 3 ) / 4] + scores[( scores.size() * 3 + 3 ) / 4] ) / 2;
    std::cout << "Min: " << min << ", LQ: " << lquart << ", Median: " << median
              << ", UQ: " << uquart << ", Max: " << max;
    std::cout << ", Mean: " << sum / (double)scores.size() << std::endl;
}

TYPED_TEST( ParameterQualityTests, FullPermutation )
{
    const uint64_t num_loops = NUM_SAMPLES;
    const uint64_t seed_start = 0xdeadbeef;
    std::vector<double> p_scores;
    for( uint64_t loop = 0; loop < num_loops; loop++ )
    {
        const uint64_t shuffle_size = 6;
        const uint64_t num_samples = SAMPLE_SIZE;

        const uint64_t num_threads = 6;
        const uint64_t samples_per_thread = ( num_samples + ( num_threads - 1 ) ) / num_threads;

        std::vector<std::unordered_map<uint64_t, uint64_t>> results_map( num_threads );
        std::vector<std::thread> threads;
        for( uint64_t tid = 0; tid < num_threads; tid++ )
        {
            threads.emplace_back( [&, tid]() {
                auto local_shuffle = this->shuffle;
                thrust::host_vector<uint64_t> input( shuffle_size );
                thrust::host_vector<uint64_t> output( shuffle_size );

                for( uint64_t i = tid * samples_per_thread;
                     i < std::min( num_samples, samples_per_thread * ( tid + 1 ) ); i++ )
                {
                    thrust::sequence( input.begin(), input.end(), 0 );
                    local_shuffle( input, output, seed_start + loop * num_samples + i, shuffle_size );
                    const uint64_t index = permutationToIndex( output, shuffle_size );
                    results_map[tid][index]++;
                }
            } );
        }

        std::unordered_map<uint64_t, uint64_t> results;
        for( auto& thread : threads )
            thread.join();

        for( auto& res : results_map )
            for( auto& pair : res )
                results[pair.first] += pair.second;

        const uint64_t size_fact = factorial( shuffle_size );
        const double expected_occurances = num_samples / (double)size_fact;

        auto& permutations = allPermutations( shuffle_size );
        double chi_squared = 0.0;
        for( uint64_t i = 0; i < size_fact; i++ )
        {
            chi_squared += pow( results[permutations[i]] - expected_occurances, 2 ) / expected_occurances;
        }

        if( USE_PVALUE )
        {
            double p_score = cephes_igamc( (double)( size_fact - 1 ) / 2.0, chi_squared / 2.0 );
            std::cout << p_score << ',' << std::flush;
            p_scores.emplace_back( p_score );
        }
        else
        {
            std::cout << chi_squared << ',' << std::flush;
        }
    }

    std::cout << std::endl;
    if( USE_PVALUE )
        reportStats( p_scores );
}

TYPED_TEST( ParameterQualityTests, PermutationLength )
{
    const uint64_t shuffle_size = SAMPLE_SIZE;
    const uint64_t num_samples = NUM_SAMPLES;
    const uint64_t max_dimension = std::min( (uint64_t)5ull, shuffle_size );

    thrust::host_vector<uint64_t> input( shuffle_size );
    thrust::sequence( input.begin(), input.end(), 0 );
    thrust::host_vector<uint64_t> output( shuffle_size );

    std::vector<std::vector<double>> p_scores( max_dimension - 2 );

    for( uint64_t i = 0; i < num_samples; i++ )
    {
        this->shuffle( input, output, i, shuffle_size );
        auto cycle_lengths = this->cycleLengths( output );
        for( uint64_t d = 2; d < max_dimension; d++ )
        {
            double chi_squared = this->sobolevaStatistic( shuffle_size, d, cycle_lengths );
            double p_value = cephes_igamc( (double)d / 2.0, chi_squared / 2.0 );
            if( USE_PVALUE )
                p_scores[d - 2].emplace_back( p_value );
            else
                p_scores[d - 2].emplace_back( chi_squared );
        }
    }


    for( auto& d_p_score : p_scores )
    {
        for( auto score : d_p_score )
            std::cout << score << ",";
        std::cout << std::endl;
        if( USE_PVALUE )
            reportStats( d_p_score );
    }
}

TYPED_TEST( ParameterQualityTests, TurningPointCount )
{
    const uint64_t shuffle_size = SAMPLE_SIZE;
    const uint64_t num_samples = NUM_SAMPLES;

    thrust::host_vector<uint64_t> input( shuffle_size );
    thrust::sequence( input.begin(), input.end(), 0 );
    thrust::host_vector<uint64_t> output( shuffle_size );

    // Distribution expected according to
    // Hombas, V. (2004). Turning Points in Random Permutations. Teaching Statistics, 26(1), 17-19.
    const double expected = 2.0 * (double)( shuffle_size - 2 ) / 3.0;
    const double stddev = sqrt( (double)( 16 * shuffle_size - 29 ) / 90.0 );

    std::vector<double> p_scores;

    for( uint64_t i = 0; i < num_samples; i++ )
    {
        this->shuffle( input, output, i, shuffle_size );
        uint64_t num_tp = 0;
        for( uint64_t j = 1; j < shuffle_size - 1; j++ )
            // Check for turning point (i.e. both neighbours bigger or smaller)
            if( ( output[j - 1] < output[j] ) == ( output[j + 1] < output[j] ) )
                num_tp++;

        const double z_score = ( (double)num_tp - expected ) / stddev;
        const double p_val = 1.0 - erf( abs( z_score ) / sqrt( 2.0 ) );
        // std::cerr << num_tp << " vs expected " << expected << " (difference of " << ((double)num_tp - expected) << "). Produced p value of: " << p_val << std::endl;
        if( USE_PVALUE )
        {
            std::cout << p_val << "," << std::flush;
            p_scores.emplace_back( p_val );
        }
        else
        {
            std::cout << z_score << "," << std::flush;
        }
    }

    std::cout << std::endl;
    if( USE_PVALUE )
        reportStats( p_scores );
}