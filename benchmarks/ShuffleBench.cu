#include "hip/hip_runtime.h"
#include "CudaHelpers.h"
#include "GatherShuffle.h"
#include "ScatterShuffle.h"
#include "ThrustInclude.h"
#include "shuffle/DartThrowing.h"
#include "shuffle/FeistelBijectiveShuffle.h"
#include "shuffle/FisherYatesShuffle.h"
#include "shuffle/LCGBijectiveShuffle.h"
#include "shuffle/MergeShuffle.h"
#include "shuffle/NoOpBijectiveShuffle.h"
#include "shuffle/RaoSandeliusShuffle.h"
#include "shuffle/StdShuffle.h"
#include <benchmark/benchmark.h>
#include <cmath>
#include <sstream>
#include <vector>

// #define HOST_BENCH 1
using DataType = uint64_t;

template <class ShuffleFunction>
static void benchmarkScatterGather( benchmark::State& state )
{
    ShuffleFunction shuffler;
    using ContainerType = typename ShuffleFunction::container_type;

    // Shuffle second param adds 0 or 1 to compare power of two (best case) vs.
    // one above power of two (worst case)
    const uint64_t num_to_shuffle = (uint64_t)state.range( 1 ) + ( 1ull << (uint64_t)state.range( 0 ) );

    ContainerType in_container( num_to_shuffle );
    ContainerType out_container( num_to_shuffle );

    // Use bijective shuffle since it is fastest and still strongly random
    FeistelBijectiveShuffle<ContainerType> temp_shuffler;
    thrust::sequence( out_container.begin(), out_container.end() );

    int seed = 0;
    for( auto _ : state )
    {
        state.PauseTiming();
        if( ( seed % 100 ) == 0 )
            temp_shuffler( out_container, in_container, seed );
#ifndef HOST_BENCH
        checkCudaError( hipDeviceSynchronize() );
#endif
        state.ResumeTiming();
        // Benchmarks raw gather speed of a random permutation
        shuffler( in_container, out_container, seed );
#ifndef HOST_BENCH
        checkCudaError( hipDeviceSynchronize() );
#endif
        seed++;
    }

    state.SetItemsProcessed( state.iterations() * num_to_shuffle );
    uint64_t log = std::log2( num_to_shuffle );
    std::stringstream s;
    s << "Shuffle 2^" << log;
    if( state.range( 1 ) )
    {
        s << " + 1";
    }
    state.SetLabel( s.str() );
}

template <class ShuffleFunction>
static void benchmarkFunction( benchmark::State& state )
{
    ShuffleFunction shuffler;
    using ContainerType = typename ShuffleFunction::container_type;

    // Shuffle second param adds 0 or 1 to compare power of two (best case) vs.
    // one above power of two (worst case)
    const uint64_t num_to_shuffle = (uint64_t)state.range( 1 ) + ( 1ull << (uint64_t)state.range( 0 ) );

    ContainerType in_container( num_to_shuffle );
    ContainerType out_container( num_to_shuffle );

    int seed = 0;
    for( auto _ : state )
    {
        shuffler( in_container, out_container, seed );
#ifndef HOST_BENCH
        checkCudaError( hipDeviceSynchronize() );
#endif
        seed++;
    }

    state.SetItemsProcessed( state.iterations() * num_to_shuffle );
    std::stringstream s;
    s << "Shuffle 2^" << state.range( 0 );
    if( state.range( 1 ) )
    {
        s << " + 1";
    }
    state.SetLabel( s.str() );
}

static void argsGenerator( benchmark::internal::Benchmark* b )
{
    // Go up by 3 so we get both odd and even numbers of bits
    std::vector<int> logs = { 8, 11, 14, 17, 20, 23, 26, 29 };
    for( int log : logs )
    {
        b->Args( { log, 0 } );
        b->Args( { log, 1 } );
    }
}

#ifndef HOST_BENCH
template <uint64_t NumRounds, class RoundFunction>
using ParamFeistelBijectiveScanShuffle =
    BijectiveFunctionScanShuffle<FeistelBijectiveFunction<NumRounds, RoundFunction>, thrust::device_vector<DataType>, DefaultRandomGenerator>;
#endif
template <uint64_t NumRounds, class RoundFunction>
using TBBParamFeistelBijectiveScanShuffle =
    BijectiveFunctionScanShuffle<FeistelBijectiveFunction<NumRounds, RoundFunction>, thrust::tbb::vector<DataType>, DefaultRandomGenerator>;

constexpr uint64_t target_num_rounds = 16;
#ifndef HOST_BENCH
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, Taus88RanluxRoundFunction<target_num_rounds, true>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, Taus88LCGRoundFunction<target_num_rounds, true>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, RanluxLCGRoundFunction<target_num_rounds, true>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, Taus88RanluxRoundFunction<target_num_rounds, false>> )
->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, Taus88LCGRoundFunction<target_num_rounds, false>> )
->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, RanluxLCGRoundFunction<target_num_rounds, false>> )
->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, WyHashRoundFunction<target_num_rounds>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    ParamFeistelBijectiveScanShuffle<target_num_rounds, RC5RoundFunction<target_num_rounds>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, LCGBijectiveScanShuffle<thrust::device_vector<DataType>> )->Apply( argsGenerator );
#endif

BENCHMARK_TEMPLATE( benchmarkFunction,
                    TBBParamFeistelBijectiveScanShuffle<target_num_rounds, Taus88RanluxRoundFunction<target_num_rounds, fast_generator>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    TBBParamFeistelBijectiveScanShuffle<target_num_rounds, Taus88LCGRoundFunction<target_num_rounds, fast_generator>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    TBBParamFeistelBijectiveScanShuffle<target_num_rounds, RanluxLCGRoundFunction<target_num_rounds, fast_generator>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    TBBParamFeistelBijectiveScanShuffle<target_num_rounds, WyHashRoundFunction<target_num_rounds>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction,
                    TBBParamFeistelBijectiveScanShuffle<target_num_rounds, RC5RoundFunction<target_num_rounds>> )
    ->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, LCGBijectiveScanShuffle<thrust::tbb::vector<DataType>> )->Apply( argsGenerator );

#ifndef HOST_BENCH
BENCHMARK_TEMPLATE( benchmarkFunction, DartThrowing<thrust::device_vector<DataType>> )->Apply( argsGenerator );
#endif

BENCHMARK_TEMPLATE( benchmarkFunction, HostDartThrowing<std::vector<DataType>, 5, 4> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, HostDartThrowing<std::vector<DataType>, 3, 2> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, HostDartThrowing<std::vector<DataType>, 2, 1> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, HostDartThrowing<std::vector<DataType>, 4, 1> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, MergeShuffle<std::vector<DataType>> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, RaoSandeliusShuffle<std::vector<DataType>> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkFunction, StdShuffle<std::vector<DataType>> )->Apply( argsGenerator );

#ifndef HOST_BENCH
BENCHMARK_TEMPLATE( benchmarkScatterGather, GatherShuffle<thrust::device_vector<DataType>> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkScatterGather, ScatterShuffle<thrust::device_vector<DataType>> )->Apply( argsGenerator );
#else
BENCHMARK_TEMPLATE( benchmarkScatterGather, GatherShuffle<thrust::host_vector<DataType>> )->Apply( argsGenerator );
BENCHMARK_TEMPLATE( benchmarkScatterGather, ScatterShuffle<thrust::host_vector<DataType>> )->Apply( argsGenerator );
#endif

BENCHMARK_MAIN();